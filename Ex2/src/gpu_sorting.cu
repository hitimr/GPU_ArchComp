# pragma once
#include <hip/hip_runtime.h>

#include "assert.h"
#include <vector>
#include <iostream>
#include <tuple>

#define BLOCK_SIZE 256 
#define GRID_SIZE 256 


//--------------------------------------
//       sort one single array
//--------------------------------------
__device__ void gpu_swap(int* vec, size_t i, size_t j){
    int temp = vec[i];
    vec[i] = vec[j];
    vec[j] = temp;
}

__global__ void gpu_even_pass(int* vec, size_t size){

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(size_t i = thread_id; i < size_t(size/2); i += num_threads){
        if (vec[2*i] > vec[2*i+1])
            gpu_swap(vec, 2*i, 2*i+1);
    }
}

__global__ void gpu_odd_pass(int* vec, size_t size){

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(size_t i = thread_id; i < size_t(size/2 - 1); i += num_threads){
        if (vec[2*i+1] > vec[2*i+2])
            gpu_swap(vec, 2*i+1, 2*i+2);
    }
}

void gpu_bubble_sort(std::vector<int> &vec){

    size_t size = vec.size(); 
    int num_bytes = vec.size() * sizeof(int);

    // allocate
    int* d_vec;
    hipMalloc((void**)&d_vec, num_bytes);

    // copy
    hipMemcpy(d_vec, vec.data(), num_bytes, hipMemcpyHostToDevice);

    // sort
    for(size_t i = 0; i < size_t(size/2); ++i){
        gpu_even_pass<<<BLOCK_SIZE, GRID_SIZE>>>(d_vec, size);
        hipDeviceSynchronize();            
        gpu_odd_pass<<<BLOCK_SIZE, GRID_SIZE>>>(d_vec, size);
        hipDeviceSynchronize();            
    }

    // copy back
    hipMemcpy(vec.data(), d_vec, sizeof(int) * vec.size(), hipMemcpyDeviceToHost);
    hipFree(d_vec);
}


//--------------------------------------
//       sort three arrays
//--------------------------------------
__global__ void gpu_even_pass_mult(int* vec, size_t size, int* v2, int* v3){

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(size_t i = thread_id; i < size_t(size/2); i += num_threads){
        if (vec[2*i] > vec[2*i+1]){
            gpu_swap(vec, 2*i, 2*i+1);
            gpu_swap(v2, 2*i, 2*i+1);
            gpu_swap(v3, 2*i, 2*i+1);
        }
    }
}


__global__ void gpu_odd_pass_mult(int* vec, size_t size, int* v2, int* v3){

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(size_t i = thread_id; i < size_t(size/2 - 1); i += num_threads){
        if (vec[2*i+1] > vec[2*i+2]){
            gpu_swap(vec, 2*i+1, 2*i+2);
            gpu_swap(v2, 2*i+1, 2*i+2);
            gpu_swap(v3, 2*i+1, 2*i+2);
        }
    }
}


void gpu_bubble_sort_mult(std::vector<int> &vec, std::vector<int> &v2, std::vector<int> &v3){

    size_t size = vec.size(); 
    int num_bytes = vec.size() * sizeof(int);

    // allocate
    int *d_vec, *d_v2, *d_v3;
    hipMalloc((void**)&d_vec, num_bytes);
    hipMalloc((void**)&d_v2, num_bytes);
    hipMalloc((void**)&d_v3, num_bytes);

    // copy
    hipMemcpy(d_vec, vec.data(), num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_v2, v2.data(), num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_v3, v3.data(), num_bytes, hipMemcpyHostToDevice);

    // sort
    for(size_t i = 0; i < size_t(size/2); ++i){
        gpu_even_pass_mult<<<BLOCK_SIZE, GRID_SIZE>>>(d_vec, size, d_v2, d_v3);
        hipDeviceSynchronize();            
        gpu_odd_pass_mult<<<BLOCK_SIZE, GRID_SIZE>>>(d_vec, size, d_v2, d_v3);
        hipDeviceSynchronize();            
    }

    // copy back
    hipMemcpy(vec.data(), d_vec, sizeof(int) * vec.size(), hipMemcpyDeviceToHost);
    hipMemcpy(v2.data(), d_v2, sizeof(int) * vec.size(), hipMemcpyDeviceToHost);
    hipMemcpy(v3.data(), d_v3, sizeof(int) * vec.size(), hipMemcpyDeviceToHost);
    hipFree(d_vec);
    hipFree(d_v2);
    hipFree(d_v3);
}



// ------------------------------------------------
//                CPU - SORTING
// ------------------------------------------------

template<typename T>
bool check_sorting(std::vector<T> &vec){
    for(size_t i = 0; i < vec.size() - 1; ++i)
        if(vec[i] > vec[i+1]) return false;
    return true;
}

template<typename T>
void swap(std::vector<T> &vec, size_t i, size_t j){
    T temp = vec[i];
    vec[i] = vec[j];
    vec[j] = temp;
}

template<typename T>
void even_pass(std::vector<T> &vec){
    for(size_t i = 0; i < size_t(vec.size()/2); ++i)
        if (vec[2*i] > vec[2*i+1])
            swap(vec, 2*i, 2*i+1);
}

template<typename T>
void odd_pass(std::vector<T> &vec){
    for(size_t i = 0; i < size_t(vec.size()/2 - 1); ++i)
        if (vec[2*i+1] > vec[2*i+2])
            swap(vec, 2*i+1, 2*i+2);
}

template<typename T>
void bubble_sort(std::vector<T> &vec){
    for(size_t i = 0; i < size_t(vec.size()/2); ++i){
        even_pass(vec);
        odd_pass(vec);
    }
}
