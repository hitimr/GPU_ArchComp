#include "common.hpp"
#include "edgelist.hpp"

void EdgeList::init_gpu() {}

void EdgeList::sync_hostToDevice()
{
  if (owner == DEVICE)
  {
    // Nothing to do
    return;
  }

  hipDeviceSynchronize();

  // coo1
  size_t bytes = sizeof(int) * coo1.size();
  hipMalloc(&d_coo1, bytes);
  hipMemcpy(d_coo1, coo1.data(), bytes, hipMemcpyHostToDevice);

  // coo2
  bytes = sizeof(int) * coo2.size();
  hipMalloc(&d_coo2, bytes);
  hipMemcpy(d_coo2, coo2.data(), bytes, hipMemcpyHostToDevice);

  // val
  bytes = sizeof(int) * val.size();
  hipMalloc(&d_val, bytes);
  hipMemcpy(d_val, val.data(), bytes, hipMemcpyHostToDevice);

  // constants
  bytes = sizeof(EdgeList);
  hipMalloc(&gpu, bytes);
  hipMemcpy(gpu, this, bytes, hipMemcpyHostToDevice);

  owner = DEVICE;
}

void EdgeList::sync_deviceToHost()
{
  if (owner == HOST)
  {
    // Nothing to do
    return;
  }

  hipDeviceSynchronize();

  // coo1
  size_t bytes = sizeof(int) * coo1.size();
  hipMemcpy(coo1.data(), d_coo1, bytes, hipMemcpyDeviceToHost);

  // coo2
  bytes = sizeof(int) * coo2.size();
  hipMemcpy(coo2.data(), d_coo2, bytes, hipMemcpyDeviceToHost);

  // val
  bytes = sizeof(int) * val.size();
  hipMemcpy(val.data(), d_val, bytes, hipMemcpyDeviceToHost);

  owner = HOST;
}

void EdgeList::reserve(size_t size)
{
  assert(size >= coo1.size());

  coo1.reserve(size);
  coo2.reserve(size);
  val.reserve(size);

  size_t bytes = size * sizeof(int);
  hipMalloc(&d_coo1, bytes);
  hipMalloc(&d_coo2, bytes);
  hipMalloc(&d_val, bytes);
}

void EdgeList::resize_and_set_num_edges(size_t size)
{
  coo1.resize(size);
  coo2.resize(size);
  val.resize(size);
  num_edges = size;

  hipFree(d_coo1);
  hipFree(d_coo2);
  hipFree(d_val);

  size_t bytes = size * sizeof(int);
  hipMalloc(&d_coo1, bytes);
  hipMalloc(&d_coo2, bytes);
  hipMalloc(&d_val, bytes);
}
