#include "common.hpp"
#include "edgelist.hpp"

void EdgeList::init_gpu() {}

void EdgeList::sync_hostToDevice()
{
  // coo1
  size_t bytes = sizeof(int) * coo1.size();
  hipMalloc(&d_coo1, bytes);
  hipMemcpy(d_coo1, coo1.data(), bytes, hipMemcpyHostToDevice);

  // coo2
  bytes = sizeof(int) * coo2.size();
  hipMalloc(&d_coo2, bytes);
  hipMemcpy(d_coo2, coo2.data(), bytes, hipMemcpyHostToDevice);

  // val
  bytes = sizeof(int) * val.size();
  hipMalloc(&d_val, bytes);
  hipMemcpy(d_val, val.data(), bytes, hipMemcpyHostToDevice);

  bytes = sizeof(EdgeList);
  hipMalloc(&gpu, bytes);
  hipMemcpy(gpu, this, bytes, hipMemcpyHostToDevice);
}

void EdgeList::sync_deviceToHost()
{
  // coo1
  size_t bytes = sizeof(int) * coo1.size();
  hipMemcpy(coo1.data(), d_coo1, bytes, hipMemcpyDeviceToHost);

  // coo2
  bytes = sizeof(int) * coo2.size();
  hipMemcpy(coo2.data(), d_coo2, bytes, hipMemcpyDeviceToHost);

  // val
  bytes = sizeof(int) * val.size();
  hipMemcpy(val.data(), d_val, bytes, hipMemcpyDeviceToHost);
}

void EdgeList::reserve(size_t size)
{
  assert(size >= coo1.size());

  coo1.reserve(size);
  coo2.reserve(size);
  val.reserve(size);

  size_t bytes = size * sizeof(int);
  hipMalloc(&d_coo1, bytes);
  hipMalloc(&d_coo2, bytes);
  hipMalloc(&d_val, bytes);
}

