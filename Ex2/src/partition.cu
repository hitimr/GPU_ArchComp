#include "hip/hip_runtime.h"
#include "common.hpp"
#include "edgelist.hpp"
#include "partition.hpp"
#include "inclusive_scan_kernel.hpp"
#include <algorithm>
#include <iostream>
#include <iterator>
#include <numeric>
#include <vector>


void partition(EdgeList &E, EdgeList &E_leq, EdgeList &E_ge, int threshold, int kernel)
{
  g_benchmarker.start("partition()");

  switch (kernel)
  {
  case PARTITION_KERNEL_CPU_NAIVE:
    partition_cpu_naive(E, E_leq, E_ge, threshold);
    break;

  case PARTITION_KERNEL_GPU:
    partition_inclusive_scan(E, E_leq, E_ge, threshold);
    break;

  default:
    throw std::invalid_argument("Unknown partition kernel");
  }

  g_benchmarker.stop("partition()");
}

void partition_cpu_naive(const EdgeList &E, EdgeList &E_leq, EdgeList &E_ge, int threshold)
{
  // allocate both to max size so vectors dont grow
  size_t max_size = E.size();
  E_leq.reserve(max_size);
  E_ge.reserve(max_size);

  for (size_t i = 0; i < E.size(); i++)
  {
    Edge e = E[i];
    if (e.weight <= threshold)
    {
      E_leq.append_edge(E[i]);
    }
    else
    {
      E_ge.append_edge(E[i]);
    }
  }
}

__global__ void check_array(int *vec, int *smaller, int *greater, size_t size, int threshold)
{

  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (size_t i = thread_id; i < size; i += num_threads)
  {
    smaller[i] = vec[i] <= threshold;
    greater[i] = vec[i] > threshold;
  }
}

__global__ void create_partitioned_array(int *values, int *start, int *target, 
                                         int *truth_values, int *scanned_values,
                                         int *new_array_values, int *new_array_start, int *new_array_target,
                                         size_t size)
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (size_t i = thread_id; i < size; i += num_threads)
  {
    if (truth_values[i]){
      new_array_values[scanned_values[i] - 1] = values[i];
      new_array_start[scanned_values[i] - 1] = start[i];
      new_array_target[scanned_values[i] - 1] = target[i];
    }
  }
}

// void partition_inclusive_scan(E, E_leq, E_big, threshold)
void partition_inclusive_scan(EdgeList &E, EdgeList &E_leq, EdgeList &E_ge, int threshold)
{
  size_t size = E.val.size();
  int num_bytes = E.val.size() * sizeof(int);

  // allocate
  int *d_E_val, *d_E_coo1, *d_E_coo2, *d_truth_small, *d_truth_big, *d_scanned_truth_small, *d_scanned_truth_big;
  hipMalloc((void **)&d_E_val, num_bytes);
  hipMalloc((void **)&d_E_coo1, num_bytes);
  hipMalloc((void **)&d_E_coo2, num_bytes);
  hipMalloc((void **)&d_truth_small, num_bytes);
  hipMalloc((void **)&d_truth_big, num_bytes);
  hipMalloc((void **)&d_scanned_truth_small, num_bytes);
  hipMalloc((void **)&d_scanned_truth_big, num_bytes);

  hipMemcpy(d_E_val, E.val.data(), num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_E_coo1, E.coo1.data(), num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_E_coo2, E.coo2.data(), num_bytes, hipMemcpyHostToDevice);

  check_array<<<GRIDSIZE, BLOCKSIZE>>>(d_E_val, d_truth_small, d_truth_big, size, threshold);

  inclusive_scan(d_truth_small, d_scanned_truth_small, size);
  inclusive_scan(d_truth_big, d_scanned_truth_big, size);

  int sum_smaller[1];
  int sum_greater[1];

  hipMemcpy(sum_smaller, d_scanned_truth_small + size - 1, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(sum_greater, d_scanned_truth_big + size - 1, sizeof(int), hipMemcpyDeviceToHost);

  int *d_E_leq_val, *d_E_leq_coo1, *d_E_leq_coo2, *d_E_ge_val, *d_E_ge_coo1, *d_E_ge_coo2;
  hipMalloc((void **)&d_E_leq_val, sizeof(int) * sum_smaller[0]);
  hipMalloc((void **)&d_E_leq_coo1, sizeof(int) * sum_smaller[0]);
  hipMalloc((void **)&d_E_leq_coo2, sizeof(int) * sum_smaller[0]);
  hipMalloc((void **)&d_E_ge_val, sizeof(int) * sum_greater[0]);
  hipMalloc((void **)&d_E_ge_coo1, sizeof(int) * sum_greater[0]);
  hipMalloc((void **)&d_E_ge_coo2, sizeof(int) * sum_greater[0]);
  
  // reserve some space here for leq and ge vectors
  E_leq.resize_and_set_num_edges(sum_smaller[0]);
  E_ge.resize_and_set_num_edges(sum_greater[0]);

  create_partitioned_array<<<GRIDSIZE, BLOCKSIZE>>>(d_E_val, d_E_coo1, d_E_coo2, d_truth_small, d_scanned_truth_small, d_E_leq_val, d_E_leq_coo1, d_E_leq_coo2, size);
  create_partitioned_array<<<GRIDSIZE, BLOCKSIZE>>>(d_E_val, d_E_coo1, d_E_coo2, d_truth_big, d_scanned_truth_big, d_E_ge_val, d_E_ge_coo1, d_E_ge_coo2, size);

  hipMemcpy(E_leq.val.data(), d_E_leq_val, sizeof(int) * sum_smaller[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_leq.coo1.data(), d_E_leq_coo1, sizeof(int) * sum_smaller[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_leq.coo2.data(), d_E_leq_coo2, sizeof(int) * sum_smaller[0], hipMemcpyDeviceToHost);
             
  hipMemcpy(E_ge.val.data(), d_E_ge_val, sizeof(int) * sum_greater[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_ge.coo1.data(), d_E_ge_coo1, sizeof(int) * sum_greater[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_ge.coo2.data(), d_E_ge_coo2, sizeof(int) * sum_greater[0], hipMemcpyDeviceToHost);

  hipFree(d_E_val);
  hipFree(d_E_coo1);
  hipFree(d_E_coo2);
  hipFree(d_E_leq_val);
  hipFree(d_E_leq_coo1);
  hipFree(d_E_leq_coo2);
  hipFree(d_E_ge_val);
  hipFree(d_E_ge_coo1);
  hipFree(d_E_ge_coo2);
  hipFree(d_truth_small);
  hipFree(d_truth_big);
  hipFree(d_scanned_truth_small);
  hipFree(d_scanned_truth_big);

}