#include "hip/hip_runtime.h"
#include "common.hpp"
#include "union_find.hpp"
#include <vector>

int find_pc(std::vector<int> &parent, int i)
{
  if (parent[i] == i)
    return i;
  else
  {
    int root = find_pc(parent, parent[i]);
    parent[i] = root;
    return root;
  }
}

// path compression on cpu
void compress_cpu_naive(std::vector<int> &parent)
{
  for (int i = 0; i < parent.size(); ++i)
  {
    // if more than one step is neccessary to find the root...
    if (parent[parent[i]] != i)
    {
      find_pc(parent, i);
    }
  }
}

// path compression on gpu
__global__ void compress_kernel(int *parent, int *result, int size)
{

  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  int pot_root;
  for (int i = thread_id; i < size; i += num_threads)
  {
    pot_root = parent[i];

    while (true)
    {
      if (parent[pot_root] == pot_root)
        break;
      else
        pot_root = parent[pot_root];
    }
    result[i] = pot_root;
  }
}

void compress_gpu(std::vector<int> &parent)
{

  size_t size = parent.size();
  int num_bytes = size * sizeof(int);

  // allocate
  int *d_parent, *d_result;
  hipMalloc((void **)&d_parent, num_bytes);
  hipMalloc((void **)&d_result, num_bytes);

  // copy
  hipMemcpy(d_parent, parent.data(), num_bytes, hipMemcpyHostToDevice);

  // compress
  compress_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_parent, d_result, size);

  // copy back
  hipMemcpy(parent.data(), d_result, num_bytes, hipMemcpyDeviceToHost);
  hipFree(d_parent);
  hipFree(d_result);
}

void UnionFind::compress(int kernel)
{
  switch (kernel)
  {
  case COMPRESS_KERNEL_CPU_NAIVE:
    compress_cpu_naive(parent);
    break;
  case COMPRESS_KERNEL_GPU:
    compress_gpu(parent);
    break;

  default:
    throw std::invalid_argument("Unknown compress kernel");
  }
}
