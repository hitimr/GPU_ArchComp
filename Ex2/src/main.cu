#include "hip/hip_runtime.h"
#include "timer.hpp"
#include <math.h>    
#include <iostream>
#include <vector>
#include <random>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#define BLOCK_SIZE 1024
#define GRID_SIZE 1024

__device__ void gpu_swap(int* vec, size_t i, size_t j){
    int temp = vec[i];
    vec[i] = vec[j];
    vec[j] = temp;
}

__global__ void gpu_even_pass(int* vec, size_t size){

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(size_t i = thread_id; i < size_t(size/2); i += num_threads){
        if (vec[2*i] > vec[2*i+1])
            gpu_swap(vec, 2*i, 2*i+1);
    }
}

__global__ void gpu_odd_pass(int* vec, size_t size){

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(size_t i = thread_id; i < size_t(size/2); i += num_threads){
        if (vec[2*i+1] > vec[2*(i+1)])
            gpu_swap(vec, 2*i+1, 2*(i+1));
    }
}

double gpu_bubble_sort(std::vector<int> &vec){

    size_t size = vec.size(); 
    int num_bytes = vec.size() * sizeof(int);

    // allocate
    int* d_vec;
    hipMalloc((void**)&d_vec, num_bytes);

    // copy
    hipMemcpy(d_vec, vec.data(), num_bytes, hipMemcpyHostToDevice);
    Timer timer;
    double time_measured = 0;
    timer.reset();
    // sort
    for(size_t i = 0; i < size_t(size/2); ++i){
        gpu_even_pass<<<BLOCK_SIZE, GRID_SIZE>>>(d_vec, size - (size % 2));
        hipDeviceSynchronize();            
        gpu_odd_pass<<<BLOCK_SIZE, GRID_SIZE>>>(d_vec, size - 1);
        hipDeviceSynchronize();            
    }
    time_measured = timer.get();

    // copy back
    hipMemcpy(vec.data(), d_vec, sizeof(int) * vec.size(), hipMemcpyDeviceToHost);
    hipFree(d_vec);

    return time_measured;
}

__global__ void gpu_merge_pass(int* vec, int* output, size_t size, size_t size_to_merge){

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    int max_scope_to_merge = size / size_to_merge;
    if( size_to_merge % size > 0) max_scope_to_merge++; 
    max_scope_to_merge = max_scope_to_merge - (max_scope_to_merge % 2);

    for(size_t i = thread_id; i < size; i += num_threads){
        int my_value = vec[i];

        int position_in_current = (i % size_to_merge);
        int scope_id = i / size_to_merge;
        
        int side = (scope_id % 2) * 2 - 1; // -1 and 1 (left / rigt)
        int merge_to_scope = scope_id - side;
        if(merge_to_scope > size / size_to_merge) continue;
        

        int size_to_iterate = min(size_to_merge, size - (merge_to_scope*size_to_merge));
        int position_in_merged = 0;

        if(side == -1){
            position_in_merged = size_to_iterate;
            for(size_t comparison_pos = 0; comparison_pos < size_to_iterate; comparison_pos++){
                if(my_value <= vec[merge_to_scope*size_to_merge + comparison_pos]){
                    position_in_merged = comparison_pos;
                    break;
                }
            }
        }
        else{
            position_in_merged = 0;
            for(size_t comparison_pos = size_to_iterate; comparison_pos > 0; comparison_pos--){
                if(my_value >= vec[merge_to_scope*size_to_merge + comparison_pos - 1]){
                    position_in_merged = comparison_pos;
                    break;
                }
            }
        }

        output[min(merge_to_scope, scope_id)*size_to_merge + position_in_current + position_in_merged] = my_value;
    }

}


double gpu_merge_sort(std::vector<int> &vec){

    size_t size = vec.size(); 
    int num_bytes = vec.size() * sizeof(int);

    // allocate
    int* d_vec;
    int* d_tmp;

    hipMalloc((void**)&d_vec, num_bytes);
    hipMalloc((void**)&d_tmp, num_bytes);
    
    
    // copy
    hipMemcpy(d_vec, vec.data(), num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_tmp, vec.data(), num_bytes, hipMemcpyHostToDevice);

    Timer timer;
    double time_measured = 0;
    timer.reset();
    // sort
    int *input = d_tmp;
    int *output = d_vec;    
    
    int *tmp;


    for(size_t size_to_merge = 1; size_to_merge < size; size_to_merge *= 2){
        tmp = input;
        input = output;
        output = tmp;


        gpu_merge_pass<<<1024, 1024>>>(input, output, size, size_to_merge);
        hipDeviceSynchronize();                     
    }
    time_measured = timer.get();

    // copy back
    hipMemcpy(vec.data(), output, sizeof(int) * vec.size(), hipMemcpyDeviceToHost);

    hipFree(d_vec);
    hipFree(d_tmp);
    return time_measured;
}

double gpu_thrust_sort_three(std::vector<int> &vec, std::vector<int> &vec1, std::vector<int> &vec2){

    size_t size = vec.size(); 
    int num_bytes = vec.size() * sizeof(int);

    thrust::host_vector<int> h_vec = vec;
    thrust::device_vector<int> d_vec = h_vec;

    thrust::host_vector<int> h_ind_vec = vec;
    thrust::device_vector<int> d_ind_vec = h_ind_vec;
    thrust::sequence(h_ind_vec.begin(), h_ind_vec.end());
    
    thrust::copy(h_ind_vec.begin(), h_ind_vec.end(), d_ind_vec.begin());

    Timer timer;
    double time_measured = 0;
    timer.reset();
    thrust::sort_by_key(d_vec.begin(), d_vec.end(), d_ind_vec.begin());
    time_measured = timer.get();

    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
    thrust::copy(h_vec.begin(), h_vec.end(), vec.begin());

    std::vector<int> indices(size);
    thrust::copy(d_ind_vec.begin(), d_ind_vec.end(), h_ind_vec.begin());
    thrust::copy(h_ind_vec.begin(), h_ind_vec.end(), indices.begin());
    

    std::vector<int> tmp_vec1, tmp_vec2;
    tmp_vec1 = vec1;
    tmp_vec2 = vec2;
    for(size_t i = 0; i < size; i++){
        vec1[i] = tmp_vec1[indices[i]];
        vec2[i] = tmp_vec2[indices[i]];
    }

    return time_measured;
}

double gpu_thrust_sort(std::vector<int> &vec){

    size_t size = vec.size(); 
    int num_bytes = vec.size() * sizeof(int);

    thrust::host_vector<int> h_vec = vec;
    thrust::device_vector<int> d_vec = h_vec;

    Timer timer;
    double time_measured = 0;
    timer.reset();
    thrust::sort(d_vec.begin(), d_vec.end());
    time_measured = timer.get();

    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
    thrust::copy(h_vec.begin(), h_vec.end(), vec.begin());

    return time_measured;
}


__global__ void gpu_merge_sort_thread_per_block_with_ind(int* input, int* output, int size, int size_to_merge, int *input_ind, int* output_ind) {

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    int number_of_patches = (size / size_to_merge) + 1;


    for(int patch_id = thread_id; patch_id < number_of_patches; patch_id += num_threads){
        int start_point = patch_id * size_to_merge;

        int mid_point = min(start_point + (size_to_merge / 2), size);
        int end_point = min(start_point + size_to_merge, size);

        int current_left_pointer = start_point;
        int current_right_pointer = mid_point;


        for (int position_in_merged = start_point; position_in_merged < end_point; position_in_merged++) {
            if ((current_left_pointer < mid_point) && 
                (current_right_pointer >= end_point || input[current_left_pointer] < input[current_right_pointer])){
                output[position_in_merged] = input[current_left_pointer];
                output_ind[position_in_merged] = input_ind[current_left_pointer];
                current_left_pointer++;
            }
            else {
                output[position_in_merged] = input[current_right_pointer];
                output_ind[position_in_merged] = input_ind[current_right_pointer];
                current_right_pointer++;
            }
        }
        
    }
}

__global__ void gpu_merge_sort_thread_per_block(int* input, int* output, int size, int size_to_merge) {

    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    int number_of_patches = (size / size_to_merge) + 1;


    for(int patch_id = thread_id; patch_id < number_of_patches; patch_id += num_threads){
        int start_point = patch_id * size_to_merge;

        int mid_point = min(start_point + (size_to_merge / 2), size);
        int end_point = min(start_point + size_to_merge, size);

        int current_left_pointer = start_point;
        int current_right_pointer = mid_point;


        for (int position_in_merged = start_point; position_in_merged < end_point; position_in_merged++) {
            if ((current_left_pointer < mid_point) && 
                (current_right_pointer >= end_point || input[current_left_pointer] < input[current_right_pointer])){
                output[position_in_merged] = input[current_left_pointer];
                current_left_pointer++;
            }
            else {
                output[position_in_merged] = input[current_right_pointer];
                current_right_pointer++;
            }
        }
        
    }
}


double improved_mergesort_three(std::vector<int> &vec, std::vector<int> &vec1, std::vector<int> &vec2) {

    int size = vec.size(); 
    int num_bytes = vec.size() * sizeof(int);

    // allocate
    int* d_vec;
    int* d_tmp;
    hipMalloc((void**)&d_vec, num_bytes);
    hipMalloc((void**)&d_tmp, num_bytes);

    // copy
    hipMemcpy(d_vec, vec.data(), num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_tmp, vec.data(), num_bytes, hipMemcpyHostToDevice);
    
    int* ind_vec;
    int* ind_tmp;

    hipMalloc((void**)&ind_vec, num_bytes);
    hipMalloc((void**)&ind_tmp, num_bytes);
    
    std::vector<int> initial(size) ;
    std::iota (std::begin(initial), std::end(initial), 0);

    hipMemcpy(ind_vec, initial.data(), num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(ind_tmp, initial.data(), num_bytes, hipMemcpyHostToDevice);
    

    Timer timer;
    double time_measured = 0;
    timer.reset();

    int *input = d_tmp;
    int *output = d_vec;

    int *input_ind = ind_tmp;
    int *output_ind = ind_vec; 

    int *tmp;
    

    timer.reset();
    bool done = false;
    for (int size_to_merge = 2; done == false; size_to_merge *= 2) {
        tmp = input;
        input = output;
        output = tmp;

        tmp = input_ind;
        input_ind = output_ind;
        output_ind = tmp;

        // Actually call the kernel
        gpu_merge_sort_thread_per_block_with_ind<<<BLOCK_SIZE, GRID_SIZE>>>(input, output, size, size_to_merge, input_ind, output_ind);
        if(size_to_merge >= size) {done = true;}
    }
    time_measured = timer.get();

    hipMemcpy(vec.data(), output, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(initial.data(), output_ind, sizeof(int) * vec.size(), hipMemcpyDeviceToHost);

    std::vector<int> tmp_vec1, tmp_vec2;
    tmp_vec1 = vec1;
    tmp_vec2 = vec2;
    for(int i = 0; i < size; i++){
        vec1[i] = tmp_vec1[initial[i]];
        vec2[i] = tmp_vec2[initial[i]];
    }
    
    // Free the GPU memory
    hipFree(d_tmp);
    hipFree(d_vec);
    hipFree(ind_tmp);
    hipFree(ind_vec);

    return time_measured;
}

double improved_mergesort(std::vector<int> &vec) {

    int size = vec.size(); 
    int num_bytes = vec.size() * sizeof(int);

    // allocate
    int* d_vec;
    int* d_tmp;
    hipMalloc((void**)&d_vec, num_bytes);
    hipMalloc((void**)&d_tmp, num_bytes);

    // copy
    hipMemcpy(d_vec, vec.data(), num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_tmp, vec.data(), num_bytes, hipMemcpyHostToDevice);
    
    Timer timer;
    double time_measured = 0;
    timer.reset();

    int *input = d_tmp;
    int *output = d_vec;
    int *tmp;
    

    timer.reset();
    bool done = false;
    for (int size_to_merge = 2; done == false; size_to_merge *= 2) {
        tmp = input;
        input = output;
        output = tmp;
        // Actually call the kernel
        gpu_merge_sort_thread_per_block<<<BLOCK_SIZE, GRID_SIZE>>>(input, output, size, size_to_merge);
        if(size_to_merge >= size) {done = true;}
    }
    hipDeviceSynchronize();  
    
    time_measured = timer.get();

    hipMemcpy(vec.data(), output, size * sizeof(int), hipMemcpyDeviceToHost);
  
    // Free the GPU memory
    hipFree(d_tmp);
    hipFree(d_vec);

    return time_measured;
}



//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//




int main()
{
    std::cout << "Hello, World" << std::endl;
    size_t N;
    size_t max_power = 30;
    size_t repetitions = 1;
    int min_range = 0;
    int max_range = 100;

    


    for(size_t i = 4; i < max_power; i++){

        Timer timer;
        double time_cpu = 0.0;
        double time_bubble = 0.0;
        double time_merge = 0.0;
        double time_thrust = 0.0;
        double time_improved_merge = 0.0;

        N = std::pow(2, i);
        
        for(size_t j = 0; j < repetitions; j++){
            std::random_device rd; // obtain a random number from hardware
            std::mt19937 gen(rd()); // seed the generator
            std::uniform_int_distribution<> distr(min_range, max_range);
        
            std::vector<int> org_vector(N);
            for(size_t i = 0; i < org_vector.size(); ++i){
                org_vector[i] = distr(gen);
            }
        
        
            std::vector<int> cpu_vector = org_vector;
        
            timer.reset();
            std::stable_sort(cpu_vector.begin(), cpu_vector.end());
            time_cpu += timer.get();

            std::vector<int> thrust_vector = org_vector;
            std::vector<int> thrust_vector1 = org_vector;
            std::vector<int> thrust_vector2 = org_vector;
        
            time_thrust += gpu_thrust_sort_three(thrust_vector, thrust_vector1, thrust_vector2);
        
        
            for(size_t i = 0; i < thrust_vector.size(); ++i){
                if(thrust_vector[i] != cpu_vector[i] || thrust_vector1[i] != cpu_vector[i] || thrust_vector2[i] != cpu_vector[i]){
                    std::cout << "ERROR IN Thrust !!!!!!!!!!!!!" << std::endl;
                    break;
                }
            }

            std::vector<int> improved_merge_vector = org_vector;
            std::vector<int> improved_merge_vector1 = org_vector;
            std::vector<int> improved_merge_vector2 = org_vector;

        
            // std::cout << "STARTING IMPROVED" << std::endl;
            time_improved_merge += improved_mergesort_three(improved_merge_vector, improved_merge_vector1, improved_merge_vector2);
        
        
            for(size_t i = 0; i < improved_merge_vector.size(); ++i){
                if(improved_merge_vector[i] != cpu_vector[i] || improved_merge_vector1[i] != cpu_vector[i] || improved_merge_vector2[i] != cpu_vector[i]){
                    std::cout << "ERROR IN improved_merge !!!!!!!!!!!!!" << std::endl;
                    break;
                }
                // std::cout << improved_merge_vector[i] << " ";
            }
            // std::cout << std::endl << " " << std::endl;


        }

        time_cpu = time_cpu / repetitions;
        time_bubble = time_bubble / repetitions;
        time_merge = time_merge / repetitions;
        time_thrust = time_thrust / repetitions;
        time_improved_merge = time_improved_merge / repetitions;

        std::cout << N << " " << time_cpu  << " " << time_thrust << " " << time_improved_merge << std::endl;
   
    
    }
    return 0;

    
}