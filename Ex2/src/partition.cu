#include "hip/hip_runtime.h"
#include "common.hpp"
#include "edgelist.hpp"
#include "partition.hpp"
#include <algorithm>
#include <iostream>
#include <iterator>
#include <numeric>
#include <vector>

void partition(EdgeList &E, EdgeList &E_leq, EdgeList &E_ge, int threshold, int kernel)
{
  g_benchmarker.start("partition()");

  switch (kernel)
  {
  case PARTITION_KERNEL_CPU_NAIVE:
    partition_cpu_naive(E, E_leq, E_ge, threshold);
    break;

  case PARTITION_KERNEL_GPU:
    partition_inclusive_scan(E, E_leq, E_ge, threshold);
    break;

  case PARTITION_KERNEL_THRUST:
    partition_thrust(E, E_leq, E_ge, threshold);
    break;

  default:
    throw std::invalid_argument("Unknown partition kernel");
  }

  g_benchmarker.stop("partition()");
}

void filter(EdgeList &E, UnionFind &P, int kernel)
{
  g_benchmarker.start("filter()");

  switch (kernel)
  {
  case FILTER_KERNEL_CPU_NAIVE:
    filter_cpu_naive(E, P);
    break;

  case FILTER_KERNEL_GPU:
    filter_gpu_naive(E, P);
    break;

  default:
    throw std::invalid_argument("Unknown filter kernel");
  }

  g_benchmarker.stop("filter()");
}

void partition_cpu_naive(const EdgeList &E, EdgeList &E_leq, EdgeList &E_ge, int threshold)
{
  // allocate both to max size so vectors dont grow
  size_t max_size = E.size();
  E_leq.reserve(max_size);
  E_ge.reserve(max_size);

  for (size_t i = 0; i < E.size(); i++)
  {
    Edge e = E[i];
    if (e.weight <= threshold)
    {
      E_leq.append_edge(E[i]);
    }
    else
    {
      E_ge.append_edge(E[i]);
    }
  }
}

// first kernel for inclusive scan
// the whole array X to be scanned gets contigously distributed among the blocks.
// every block performs an inclusive scan on its array and writes the results to Y.
// Y then is partly inclusively scanned. what is still missing, is adding the end value of block
// i-1 to all elements in block i. This will later happen in a second kernel.
// all the blockoffsets get stored in carries.
__global__ void scan_kernel_1(int const *X, int *Y, int N, int *carries)
{
  __shared__ double shared_buffer[BLOCKSIZE];
  int my_value;

  unsigned int work_per_thread = (N - 1) / (gridDim.x * blockDim.x) + 1;
  unsigned int block_start = work_per_thread * blockDim.x * blockIdx.x;
  unsigned int block_stop = work_per_thread * blockDim.x * (blockIdx.x + 1);
  unsigned int block_offset = 0;

  // run scan on each section, this for loop is necessary if there are more elements in the array
  // than there are threads in total.
  for (unsigned int i = block_start + threadIdx.x; i < block_stop; i += blockDim.x)
  {
    // load data:
    my_value = (i < N) ? X[i] : 0;

    // inclusive scan in shared buffer:
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
      __syncthreads();
      shared_buffer[threadIdx.x] = my_value;
      __syncthreads();
      if (threadIdx.x >= stride)
        my_value += shared_buffer[threadIdx.x - stride];
    }
    __syncthreads();
    shared_buffer[threadIdx.x] = my_value;
    __syncthreads();

    // write to output array
    if (i < N)
      Y[i] = block_offset + my_value;

    block_offset += shared_buffer[blockDim.x - 1];
  }

  // write carry:
  if (threadIdx.x == 0)
    carries[blockIdx.x] = block_offset;
}

// Y is partly inclusively scanned. Here the offsets of each block get added.
__global__ void scan_kernel_2(int *Y, int N, int const *carries)
{
  unsigned int work_per_thread = (N - 1) / (gridDim.x * blockDim.x) + 1;
  unsigned int block_start = work_per_thread * blockDim.x * blockIdx.x;
  unsigned int block_stop = work_per_thread * blockDim.x * (blockIdx.x + 1);

  __shared__ int shared_offset;
  __shared__ int shared_buffer[GRIDSIZE];

  // load data:
  int my_carry = carries[threadIdx.x];

  // inclusive scan in the carries array
  for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
  {
    __syncthreads();
    shared_buffer[threadIdx.x] = my_carry;
    __syncthreads();
    if (threadIdx.x >= stride)
      my_carry += shared_buffer[threadIdx.x - stride];
  }
  __syncthreads();
  shared_buffer[threadIdx.x] = my_carry;
  __syncthreads();

  if (threadIdx.x == 0)
    shared_offset = (blockIdx.x > 0) ? shared_buffer[blockIdx.x - 1] : 0;
  ;

  __syncthreads();

  // add offset to each element in the block:
  for (unsigned int i = block_start + threadIdx.x; i < block_stop; i += blockDim.x)
    if (i < N)
      Y[i] += shared_offset;
}

// The general structure of the inclusive scan and most parts in the respective kernel codes are
// taken from Dr. Rupp's lecture "Computational Sciende on Many Core Architectures". We simplified
// it a bit to be restricted to work when GRIDSIZE == BLOCKSIZE. That saves us launching one
// intermediate kernel to perform an inclusive scan on the carries array.
void inclusive_scan(int const *input, int *output, int N)
{

  int *carries;
  hipMalloc(&carries, sizeof(int) * GRIDSIZE);

  // First step: Scan within each thread group and write carries
  scan_kernel_1<<<GRIDSIZE, BLOCKSIZE>>>(input, output, N, carries);

  hipDeviceSynchronize();

  // Second step: Offset each thread group accordingly
  scan_kernel_2<<<GRIDSIZE, BLOCKSIZE>>>(output, N, carries);

  hipDeviceSynchronize();

  hipFree(carries);
}

__global__ void check_array(int *vec, int *smaller, int *greater, size_t size, int threshold)
{

  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (size_t i = thread_id; i < size; i += num_threads)
  {
    smaller[i] = vec[i] <= threshold;
    greater[i] = vec[i] > threshold;
  }
}

__global__ void create_partitioned_array(int *values, int *start, int *target, int *truth_values,
                                         int *scanned_values, int *new_array_values,
                                         int *new_array_start, int *new_array_target, size_t size)
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (size_t i = thread_id; i < size; i += num_threads)
  {
    if (truth_values[i])
    {
      new_array_values[scanned_values[i] - 1] = values[i];
      new_array_start[scanned_values[i] - 1] = start[i];
      new_array_target[scanned_values[i] - 1] = target[i];
    }
  }
}

// void partition_inclusive_scan(E, E_leq, E_big, threshold)
void partition_inclusive_scan(EdgeList &E, EdgeList &E_leq, EdgeList &E_ge, int threshold)
{
  size_t size = E.val.size();
  int num_bytes = E.val.size() * sizeof(int);

  // allocate
  int *d_E_val, *d_E_coo1, *d_E_coo2, *d_truth_small, *d_truth_big, *d_scanned_truth_small,
      *d_scanned_truth_big;
  hipMalloc((void **)&d_E_val, num_bytes);
  hipMalloc((void **)&d_E_coo1, num_bytes);
  hipMalloc((void **)&d_E_coo2, num_bytes);
  hipMalloc((void **)&d_truth_small, num_bytes);
  hipMalloc((void **)&d_truth_big, num_bytes);
  hipMalloc((void **)&d_scanned_truth_small, num_bytes);
  hipMalloc((void **)&d_scanned_truth_big, num_bytes);

  hipMemcpy(d_E_val, E.val.data(), num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_E_coo1, E.coo1.data(), num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_E_coo2, E.coo2.data(), num_bytes, hipMemcpyHostToDevice);

  check_array<<<GRIDSIZE, BLOCKSIZE>>>(d_E_val, d_truth_small, d_truth_big, size, threshold);

  inclusive_scan(d_truth_small, d_scanned_truth_small, size);
  inclusive_scan(d_truth_big, d_scanned_truth_big, size);

  int sum_smaller[1];
  int sum_greater[1];

  hipMemcpy(sum_smaller, d_scanned_truth_small + size - 1, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(sum_greater, d_scanned_truth_big + size - 1, sizeof(int), hipMemcpyDeviceToHost);

  int *d_E_leq_val, *d_E_leq_coo1, *d_E_leq_coo2, *d_E_ge_val, *d_E_ge_coo1, *d_E_ge_coo2;
  hipMalloc((void **)&d_E_leq_val, sizeof(int) * sum_smaller[0]);
  hipMalloc((void **)&d_E_leq_coo1, sizeof(int) * sum_smaller[0]);
  hipMalloc((void **)&d_E_leq_coo2, sizeof(int) * sum_smaller[0]);
  hipMalloc((void **)&d_E_ge_val, sizeof(int) * sum_greater[0]);
  hipMalloc((void **)&d_E_ge_coo1, sizeof(int) * sum_greater[0]);
  hipMalloc((void **)&d_E_ge_coo2, sizeof(int) * sum_greater[0]);

  // reserve some space here for leq and ge vectors
  E_leq.resize_and_set_num_edges(sum_smaller[0]);
  E_ge.resize_and_set_num_edges(sum_greater[0]);

  create_partitioned_array<<<GRIDSIZE, BLOCKSIZE>>>(d_E_val, d_E_coo1, d_E_coo2, d_truth_small,
                                                    d_scanned_truth_small, d_E_leq_val,
                                                    d_E_leq_coo1, d_E_leq_coo2, size);
  create_partitioned_array<<<GRIDSIZE, BLOCKSIZE>>>(d_E_val, d_E_coo1, d_E_coo2, d_truth_big,
                                                    d_scanned_truth_big, d_E_ge_val, d_E_ge_coo1,
                                                    d_E_ge_coo2, size);

  hipMemcpy(E_leq.val.data(), d_E_leq_val, sizeof(int) * sum_smaller[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_leq.coo1.data(), d_E_leq_coo1, sizeof(int) * sum_smaller[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_leq.coo2.data(), d_E_leq_coo2, sizeof(int) * sum_smaller[0], hipMemcpyDeviceToHost);

  hipMemcpy(E_ge.val.data(), d_E_ge_val, sizeof(int) * sum_greater[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_ge.coo1.data(), d_E_ge_coo1, sizeof(int) * sum_greater[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_ge.coo2.data(), d_E_ge_coo2, sizeof(int) * sum_greater[0], hipMemcpyDeviceToHost);

  hipFree(d_E_val);
  hipFree(d_E_coo1);
  hipFree(d_E_coo2);
  hipFree(d_E_leq_val);
  hipFree(d_E_leq_coo1);
  hipFree(d_E_leq_coo2);
  hipFree(d_E_ge_val);
  hipFree(d_E_ge_coo1);
  hipFree(d_E_ge_coo2);
  hipFree(d_truth_small);
  hipFree(d_truth_big);
  hipFree(d_scanned_truth_small);
  hipFree(d_scanned_truth_big);
}

// condition for partitioning with thrust
struct is_less_equal
{
  int threshold;
  is_less_equal(int t): threshold(t) {}

  __host__ __device__
  bool operator()(const int &x)
  {
    return x < threshold;
  }
};


void partition_thrust(EdgeList &E, EdgeList &E_leq, EdgeList &E_ge, int threshold){
  
  size_t size = E.val.size(); 
  int num_bytes = E.val.size() * sizeof(int);

  thrust::host_vector<int> h_vec = E.val;
  thrust::device_vector<int> d_vec = h_vec;

  thrust::host_vector<int> h_ind_vec = E.val;
  thrust::device_vector<int> d_ind_vec = h_ind_vec;
  thrust::sequence(h_ind_vec.begin(), h_ind_vec.end());
    
  thrust::copy(h_ind_vec.begin(), h_ind_vec.end(), d_ind_vec.begin());

  thrust::host_vector<int> h_vec_ge = E_ge.val;
  thrust::device_vector<int> d_vec_ge = h_vec_ge;
  
  auto middle = thrust::stable_partition(thrust::device, d_vec.begin(), d_vec.end(), is_less_equal(threshold));
  
  thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
  thrust::copy(h_vec.begin(), h_vec.end(), E.val.begin());

  std::vector<int> indices(size);
  thrust::copy(d_ind_vec.begin(), d_ind_vec.end(), h_ind_vec.begin());
  thrust::copy(h_ind_vec.begin(), h_ind_vec.end(), indices.begin());
    

  std::vector<int> tmp_vec1, tmp_vec2;
  tmp_vec1 = E.coo1;
  tmp_vec2 = E.coo2;
  for(size_t i = 0; i < size; i++){
      E.coo1[i] = tmp_vec1[indices[i]];
      E.coo2[i] = tmp_vec2[indices[i]];
  }

  int length_smaller_array = middle - d_vec.begin();

  E_leq.resize_and_set_num_edges(length_smaller_array-1);
  E_ge.resize_and_set_num_edges(length_smaller_array);

  thrust::copy(E.val.begin(), E.val.begin() + length_smaller_array-1, E_leq.val.begin());
  thrust::copy(E.coo1.begin(), E.coo1.begin() + length_smaller_array-1, E_leq.coo1.begin());
  thrust::copy(E.coo2.begin(), E.coo2.begin() + length_smaller_array-1, E_leq.coo2.begin());

  thrust::copy(E.val.begin() + length_smaller_array, E.val.end(), E_ge.val.begin());
  thrust::copy(E.coo1.begin() + length_smaller_array, E.coo1.end(), E_ge.coo1.begin());
  thrust::copy(E.coo2.begin() + length_smaller_array, E.coo2.end(), E_ge.coo2.begin());

}

void filter_cpu_naive(EdgeList &E, UnionFind &P)
{
  EdgeList E_filt;
  E_filt.reserve(E.size());

  for (size_t i = 0; i < E.size(); i++)
  {
    Edge e = E[i];
    if (P.find(e.source) != P.find(e.target))
    {
      E_filt.append_edge(e);
    }
  }

  E = E_filt;
}

__device__ int find_gpu(int i, int *group_array)
{
  if (group_array[i] == i)
    return i;
  else
    return find_gpu(group_array[i], group_array);
}

__global__ void check_array_filter(int *group_array, int *coo1, int *coo2, int *truth_array,
                                   int size)
{

  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (size_t i = thread_id; i < size; i += num_threads)
  {
    truth_array[i] = (find_gpu(coo1[i], group_array) != find_gpu(coo2[i], group_array));
  }
}

__global__ void create_partitioned_array_filter(int *values, int *start, int *target,
                                                int *truth_values, int *scanned_values,
                                                int *new_array_values, int *new_array_start,
                                                int *new_array_target, size_t size)
{
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (size_t i = thread_id; i < size; i += num_threads)
  {
    if (truth_values[i])
    {
      new_array_values[scanned_values[i] - 1] = values[i];
      new_array_start[scanned_values[i] - 1] = start[i];
      new_array_target[scanned_values[i] - 1] = target[i];
    }
  }
}

void filter_gpu_naive(EdgeList &E, UnionFind &P)
{
  EdgeList E_new;

  size_t size = E.val.size();
  int num_bytes = E.val.size() * sizeof(int);

  int num_bytes_parents = P.parent.size() * sizeof(int);

  // allocate
  int *d_E_val, *d_E_coo1, *d_E_coo2, *d_truth, *d_scanned_truth, *d_parents;
  hipMalloc((void **)&d_E_val, num_bytes);
  hipMalloc((void **)&d_E_coo1, num_bytes);
  hipMalloc((void **)&d_E_coo2, num_bytes);
  hipMalloc((void **)&d_truth, num_bytes);
  hipMalloc((void **)&d_scanned_truth, num_bytes);
  hipMalloc((void **)&d_parents, num_bytes_parents);

  hipMemcpy(d_E_val, E.val.data(), num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_E_coo1, E.coo1.data(), num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_E_coo2, E.coo2.data(), num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_parents, P.parent.data(), num_bytes_parents, hipMemcpyHostToDevice);

  check_array_filter<<<GRIDSIZE, BLOCKSIZE>>>(d_parents, d_E_coo1, d_E_coo2, d_truth, size);

  inclusive_scan(d_truth, d_scanned_truth, size);

  int sum[1];

  hipMemcpy(sum, d_scanned_truth + size - 1, sizeof(int), hipMemcpyDeviceToHost);

  int *d_E_new_val, *d_E_new_coo1, *d_E_new_coo2;
  hipMalloc((void **)&d_E_new_val, sizeof(int) * sum[0]);
  hipMalloc((void **)&d_E_new_coo1, sizeof(int) * sum[0]);
  hipMalloc((void **)&d_E_new_coo2, sizeof(int) * sum[0]);

  // reserve some space here for leq and ge vectors
  E_new.resize_and_set_num_edges(sum[0]);

  create_partitioned_array_filter<<<GRIDSIZE, BLOCKSIZE>>>(d_E_val, d_E_coo1, d_E_coo2, d_truth,
                                                           d_scanned_truth, d_E_new_val,
                                                           d_E_new_coo1, d_E_new_coo2, size);

  hipMemcpy(E_new.val.data(), d_E_new_val, sizeof(int) * sum[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_new.coo1.data(), d_E_new_coo1, sizeof(int) * sum[0], hipMemcpyDeviceToHost);
  hipMemcpy(E_new.coo2.data(), d_E_new_coo2, sizeof(int) * sum[0], hipMemcpyDeviceToHost);

  hipFree(d_E_val);
  hipFree(d_E_coo1);
  hipFree(d_E_coo2);
  hipFree(d_E_new_val);
  hipFree(d_E_new_coo1);
  hipFree(d_E_new_coo2);
  hipFree(d_truth);
  hipFree(d_scanned_truth);

  E = E_new;
}