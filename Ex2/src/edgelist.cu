#include "common.hpp"
#include "edgelist.hpp"

void EdgeList::init_gpu() {}

void EdgeList::sync_hostToDevice()
{
  if (owner == DEVICE)
  {
    // Nothing to do
    return;
  }

  hipDeviceSynchronize();

  hipFree(d_coo1);
  hipFree(d_coo2);
  hipFree(d_val);

  // coo1
  size_t bytes = sizeof(int) * size();
  hipMalloc(&d_coo1, bytes);
  hipMemcpy(d_coo1, coo1, bytes, hipMemcpyHostToDevice);

  // coo2
  bytes = sizeof(int) * size();
  hipMalloc(&d_coo2, bytes);
  hipMemcpy(d_coo2, coo2, bytes, hipMemcpyHostToDevice);

  // val
  bytes = sizeof(int) * size();
  hipMalloc(&d_val, bytes);
  hipMemcpy(d_val, val, bytes, hipMemcpyHostToDevice);

  owner = DEVICE;
}

void EdgeList::sync_deviceToHost()
{
  if (owner == HOST)
  {
    // Nothing to do
    return;
  }

  hipDeviceSynchronize();

  // coo1
  size_t bytes = sizeof(int) * size();
  hipMemcpy(coo1, d_coo1, bytes, hipMemcpyDeviceToHost);

  // coo2
  bytes = sizeof(int) * size();
  hipMemcpy(coo2, d_coo2, bytes, hipMemcpyDeviceToHost);

  // val
  bytes = sizeof(int) * size();
  hipMemcpy(val, d_val, bytes, hipMemcpyDeviceToHost);
  
  owner = HOST;
}

void EdgeList::reserve(size_t new_size)
{
  assert(new_size >= size());

  // reserve on CPU
  size_t bytes = new_size * sizeof(int);
  if (g_options.count("pinned-memory"))
  {
    // Use pinned memory
    hipHostMalloc(&val, bytes);
    hipHostMalloc(&coo1, bytes);
    hipHostMalloc(&coo2, bytes);
  }
  else
  {
    // use regular memory
    val = new int[new_size];
    coo1 = new int[new_size];
    coo2 = new int[new_size];
  }

  // Reserve on GPU
  hipMalloc(&d_coo1, bytes);
  hipMalloc(&d_coo2, bytes);
  hipMalloc(&d_val, bytes);
}

void EdgeList::resize_and_set_num_edges(size_t size)
{
  // reserve on CPU
  size_t bytes = size * sizeof(int);
  if (g_options.count("pinned-memory"))
  {
    // Use pinned memory
    hipHostMalloc(&val, bytes);
    hipHostMalloc(&coo1, bytes);
    hipHostMalloc(&coo2, bytes);
  }
  else
  {
    // use regular memory
    val = new int[size];
    coo1 = new int[size];
    coo2 = new int[size];
  }

  num_edges = size;

  hipFree(d_coo1);
  hipFree(d_coo2);
  hipFree(d_val);

  hipMalloc(&d_coo1, bytes);
  hipMalloc(&d_coo2, bytes);
  hipMalloc(&d_val, bytes);
}
